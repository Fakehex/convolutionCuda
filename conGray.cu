#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>
#include <omp.h>

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void grayscale( unsigned char * rgb, unsigned char * g, std::size_t cols, std::size_t rows ) {
  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  auto j = blockIdx.y * blockDim.y + threadIdx.y;
  if( i < cols && j < rows ) {
    g[ j * cols + i ] = (
			 307 * rgb[ 3 * ( j * cols + i ) ]
			 + 604 * rgb[ 3 * ( j * cols + i ) + 1 ]
			 + 113 * rgb[  3 * ( j * cols + i ) + 2 ]
			 ) / 1024;
  }
}

__global__ void convolution_global_memory_gray(unsigned char *N,float *M,unsigned char* g,std::size_t cols, std::size_t rows,std::size_t mask_size){
  int paddingSize = ( mask_size-1 )/2;
  unsigned int paddedH = cols + 2 * paddingSize;
  unsigned int paddedW = rows + 2 * paddingSize;

  int i = blockIdx.x * blockDim.x + threadIdx.x + paddingSize;
  int j = blockIdx.y * blockDim.y + threadIdx.y + paddingSize;

  if( (j >= paddingSize) && (j < paddedW-paddingSize) && (i >= paddingSize) && (i<paddedH-paddingSize)) {
    unsigned int oPixelPos = (j - paddingSize ) * cols + (i -paddingSize);
    for(int k = -paddingSize; k <= paddingSize; k++){
      for(int l = -paddingSize; l<=paddingSize; l++){
        unsigned int iPixelPos = (j+l)*cols+(i+k);
        unsigned int coefPos = (k + paddingSize) * mask_size + (l+ paddingSize);
        g[oPixelPos] += N[iPixelPos] * M[coefPos];
      }
    }
  }
}
//filtre
static void simple_blur(std::vector< float >  &M_h, int mask_size){
  for(int i = 0; i< mask_size; i++){
    for(int j = 0; j< mask_size; j++){
      M_h[i+j*mask_size] = 1.0/(mask_size*mask_size);
    }
  }
}
//filtre de mask_size=3 attention dans le main a son utilisation
static void left_sobel_maskSize3(std::vector< float >  &M_h){
  unsigned int mask_size = 3;
  for(int i = 0; i< mask_size; i++){
    for(int j = 0; j< mask_size; j++){
      if(i==1){
        M_h[i+j*mask_size] = 0;
      }else{
        if(i==0){
          if(j==1){
            M_h[i+j*mask_size] = 2;
          }else{
            M_h[i+j*mask_size] = 1;
          }
        }
        if(i==2){
          if(j==1){
            M_h[i+j*mask_size] = -2.0;
          }else{
            M_h[i+j*mask_size] = -1.0;

          }
        }

      }
    }
  }
}
int main()
{

  cv::Mat m_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );
  auto rgb = m_in.data;
  auto rows = m_in.rows;
  auto cols = m_in.cols;
  //g sortie du grayscale
  //g resultat de la convolution
  std::vector< unsigned char > g( rows * cols );
  std::vector< unsigned char > g2( rows * cols );
  unsigned char * rgb_d;
  unsigned char * g_d;
  unsigned char * g2_d;
  unsigned char * out_d;


  unsigned int mask_size = 9;
  //M_h est notre filtre sur le host
  std::vector< float > M_h( mask_size * mask_size * sizeof(float) );
  //simple blur
  simple_blur(M_h,mask_size);
  //left_Sobel seulement avec mask 3
  //left_sobel_maskSize3(M_h);

  // PaddingSize est l'ecart suplementaire qu'il faut avoir sur chaque bord de l'image
  //PaddedW -> largeur de l'image avec padding
  //PaddedH -> longeur de l'image avec padding
  //data_pad -> notre image avec padding sur host
  std::size_t paddingSize = (mask_size-1)/2;
  std::size_t paddedW = rows + 2 * paddingSize;
  std::size_t paddedH = cols + 2 * paddingSize;
  std::vector< unsigned char > data_pad( paddedH * paddedW );

  // GRAYSACLE DEBUT
  HANDLE_ERROR(hipMalloc( &rgb_d, 3 * rows * cols ));
  HANDLE_ERROR(hipMalloc( &g_d, rows * cols ));
  HANDLE_ERROR(hipMalloc( &g2_d, rows * cols ));
  HANDLE_ERROR(hipMalloc( &out_d, rows * cols ));
  HANDLE_ERROR(hipMemcpy( rgb_d, rgb, 3 * rows * cols, hipMemcpyHostToDevice ));
  dim3 t( 32, 32 );
  dim3 b( ( cols - 1) / t.x + 1 , ( rows - 1 ) / t.y + 1 );

  grayscale<<< b, t >>>( rgb_d, g_d, cols, rows );

  HANDLE_ERROR(hipMemcpy( g.data(), g_d, rows * cols, hipMemcpyDeviceToHost ));
  //GRAYSCALE FIN -> g est notre image grise

  //creation de l'image avec le padding dans data_pad
  for(int i=0; i < paddedW ; i++){
    for(int j=0; j < paddedH ; j++){
      if((i<=paddingSize && j<=paddingSize)|| (i>=paddedW-paddingSize & j>=paddedH-paddingSize)){
        data_pad[i+j*paddedW] = 255;
      }else{
        data_pad[i+j*paddedW] = g[i+j*paddedW];
      }
    }
  }
  float * M_d;
  unsigned char * data_d;
  // On alloue de la place pour le filtre et notre image avec padding
  HANDLE_ERROR(hipMalloc( &M_d, mask_size * mask_size * sizeof(float)));
  HANDLE_ERROR(hipMalloc( &data_d, paddedH * paddedW ));

  //On copie du cpu vers gpu image et filtre
  HANDLE_ERROR(hipMemcpy(data_d, data_pad.data(), paddedW * paddedH, hipMemcpyHostToDevice ));
  HANDLE_ERROR(hipMemcpy(M_d, M_h.data(),mask_size * mask_size*sizeof(float),hipMemcpyHostToDevice));


  dim3 b2( ( paddedH - 1) / t.x + 1 , ( paddedW - 1 ) / t.y + 1 );

  //On execute le kernel
  convolution_global_memory_gray<<< b2, t >>>( data_d,M_d, g2_d, cols, rows,mask_size );

  //copie le résultat du Gpu sur cpu
  HANDLE_ERROR(hipMemcpy( g2.data(), g2_d, rows * cols, hipMemcpyDeviceToHost ));

  cv::Mat m_out( rows, cols, CV_8UC1, g2.data() );
  cv::imwrite( "out.jpg", m_out );

  hipDeviceSynchronize();

  hipFree( rgb_d);
  hipFree( g_d);
  return 0;
}
